#include "hip/hip_runtime.h"
/**
 * 文件名: convolution_cuda.cu
 * 描述: 使用CUDA实现的卷积计算
 * 
 * 卷积原理:
 * 卷积是信号处理和图像处理中的基本操作。在离散情况下，它表示为：
 * (f * g)[n] = Σ f[m] * g[n-m]
 * 
 * 在图像处理中，二维卷积公式为:
 * O[i,j] = Σ Σ I[i+m,j+n] * K[m,n]
 * 
 * CUDA并行化思路:
 * 1. 每个CUDA线程负责计算输出矩阵的一个元素
 * 2. 利用GPU的大规模并行计算能力加速卷积运算
 * 3. 对共享内存和全局内存访问进行优化
 */

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <iomanip>

// CUDA错误检查宏
#define CUDA_CHECK_ERROR(err) \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s, at line %d\n", hipGetErrorString(err), __LINE__); \
        exit(EXIT_FAILURE); \
    }

// 设备端卷积计算核函数 - 基础版本
__global__ void convolutionKernel(float* input, float* kernel, float* output, 
                                  int inputRows, int inputCols, 
                                  int kernelRows, int kernelCols,
                                  int outputRows, int outputCols) {
    // 计算当前线程负责的输出矩阵位置
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // 确保在输出矩阵范围内
    if (row < outputRows && col < outputCols) {
        float sum = 0.0f;
        
        // 应用卷积核
        for (int ki = 0; ki < kernelRows; ki++) {
            for (int kj = 0; kj < kernelCols; kj++) {
                int inputRow = row + ki;
                int inputCol = col + kj;
                sum += input[inputRow * inputCols + inputCol] * kernel[ki * kernelCols + kj];
            }
        }
        
        // 写入结果
        output[row * outputCols + col] = sum;
    }
}

// 设备端卷积计算核函数 - 使用共享内存的优化版本
__global__ void convolutionKernelShared(float* input, float* kernel, float* output, 
                                        int inputRows, int inputCols, 
                                        int kernelRows, int kernelCols,
                                        int outputRows, int outputCols) {
    extern __shared__ float sharedKernel[];
    
    // 计算当前线程负责的输出矩阵位置
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // 将卷积核加载到共享内存中 - 所有线程协作完成
    if (threadIdx.y < kernelRows && threadIdx.x < kernelCols) {
        sharedKernel[threadIdx.y * kernelCols + threadIdx.x] = kernel[threadIdx.y * kernelCols + threadIdx.x];
    }
    
    // 确保所有线程都加载完成
    __syncthreads();
    
    // 确保在输出矩阵范围内
    if (row < outputRows && col < outputCols) {
        float sum = 0.0f;
        
        // 应用卷积核 - 从共享内存读取卷积核数据
        for (int ki = 0; ki < kernelRows; ki++) {
            for (int kj = 0; kj < kernelCols; kj++) {
                int inputRow = row + ki;
                int inputCol = col + kj;
                sum += input[inputRow * inputCols + inputCol] * sharedKernel[ki * kernelCols + kj];
            }
        }
        
        // 写入结果
        output[row * outputCols + col] = sum;
    }
}

// 为矩阵分配CPU内存
float* allocate_matrix_cpu(int rows, int cols) {
    float* matrix = (float*)malloc(rows * cols * sizeof(float));
    if (!matrix) {
        fprintf(stderr, "CPU memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
    return matrix;
}

// 为矩阵分配GPU内存
float* allocate_matrix_gpu(int rows, int cols) {
    float* dev_matrix;
    hipError_t err = hipMalloc((void**)&dev_matrix, rows * cols * sizeof(float));
    CUDA_CHECK_ERROR(err);
    return dev_matrix;
}

// 创建随机矩阵
void create_random_matrix(float* matrix, int rows, int cols, float min_val, float max_val) {
    // 初始化随机数生成器(仅初始化一次，避免快速重复调用导致相同序列)
    static int seeded = 0;
    if (!seeded) {
        srand((unsigned int)time(NULL));
        seeded = 1;
    }
    
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            // 生成范围在[min_val, max_val]的随机浮点数
            float random_val = min_val + ((float)rand() / RAND_MAX) * (max_val - min_val);
            matrix[i * cols + j] = random_val;
        }
    }
}

// 打印矩阵
void print_matrix(float* matrix, int rows, int cols, const char* name) {
    printf("%s (%dx%d):\n", name, rows, cols);
    
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%6.2f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// 使用CPU执行卷积计算(用于结果比较)
void convolve_cpu(float* input, float* kernel, float* output,
                 int inputRows, int inputCols,
                 int kernelRows, int kernelCols) {
    // 计算输出矩阵的尺寸
    int outputRows = inputRows - kernelRows + 1;
    int outputCols = inputCols - kernelCols + 1;
    
    // 计算卷积
    for (int i = 0; i < outputRows; i++) {
        for (int j = 0; j < outputCols; j++) {
            // 对当前位置应用卷积核
            float sum = 0.0f;
            for (int ki = 0; ki < kernelRows; ki++) {
                for (int kj = 0; kj < kernelCols; kj++) {
                    sum += input[(i + ki) * inputCols + (j + kj)] * kernel[ki * kernelCols + kj];
                }
            }
            output[i * outputCols + j] = sum;
        }
    }
}

// 使用CUDA执行卷积计算
void convolve_cuda(float* h_input, float* h_kernel, float* h_output,
                  int inputRows, int inputCols,
                  int kernelRows, int kernelCols,
                  bool useSharedMemory = false) {
    // 计算输出矩阵的尺寸
    int outputRows = inputRows - kernelRows + 1;
    int outputCols = inputCols - kernelCols + 1;
    
    // 分配设备内存
    float* d_input = allocate_matrix_gpu(inputRows, inputCols);
    float* d_kernel = allocate_matrix_gpu(kernelRows, kernelCols);
    float* d_output = allocate_matrix_gpu(outputRows, outputCols);
    
    // 将输入数据从主机内存复制到设备内存
    hipError_t err = hipMemcpy(d_input, h_input, inputRows * inputCols * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR(err);
    
    err = hipMemcpy(d_kernel, h_kernel, kernelRows * kernelCols * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR(err);
    
    // 定义CUDA线程块和网格大小
    dim3 blockSize(16, 16);
    dim3 gridSize((outputCols + blockSize.x - 1) / blockSize.x, 
                  (outputRows + blockSize.y - 1) / blockSize.y);
    
    // 根据参数选择使用基础版本还是共享内存优化版本
    if (useSharedMemory) {
        // 计算共享内存大小 - 用于存储卷积核
        int sharedMemSize = kernelRows * kernelCols * sizeof(float);
        
        // 启动CUDA核函数 - 共享内存版本
        convolutionKernelShared<<<gridSize, blockSize, sharedMemSize>>>(
            d_input, d_kernel, d_output,
            inputRows, inputCols,
            kernelRows, kernelCols,
            outputRows, outputCols
        );
    } else {
        // 启动CUDA核函数 - 基础版本
        convolutionKernel<<<gridSize, blockSize>>>(
            d_input, d_kernel, d_output,
            inputRows, inputCols,
            kernelRows, kernelCols,
            outputRows, outputCols
        );
    }
    
    // 检查内核执行错误
    err = hipGetLastError();
    CUDA_CHECK_ERROR(err);
    
    // 同步设备，确保计算完成
    err = hipDeviceSynchronize();
    CUDA_CHECK_ERROR(err);
    
    // 将结果从设备内存复制回主机内存
    err = hipMemcpy(h_output, d_output, outputRows * outputCols * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR(err);
    
    // 释放设备内存
    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
}

// 创建边缘检测卷积核
void create_edge_detection_kernel(float* kernel) {
    // 拉普拉斯算子卷积核
    float laplacian[9] = {
        -1, -1, -1,
        -1,  8, -1,
        -1, -1, -1
    };
    
    for (int i = 0; i < 9; i++) {
        kernel[i] = laplacian[i];
    }
}

// 创建高斯平滑卷积核
void create_gaussian_kernel(float* kernel, int size, float sigma) {
    float sum = 0.0f;
    int center = size / 2;
    
    // 计算高斯函数值
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            int x = i - center;
            int y = j - center;
            // 高斯函数：G(x,y) = (1/(2*pi*sigma^2)) * e^(-(x^2+y^2)/(2*sigma^2))
            float value = exp(-(x*x + y*y) / (2 * sigma * sigma)) / (2 * 3.14159f * sigma * sigma);
            kernel[i * size + j] = value;
            sum += value;
        }
    }
    
    // 归一化卷积核（确保所有元素和为1）
    for (int i = 0; i < size * size; i++) {
        kernel[i] /= sum;
    }
}

// 计算两个矩阵之间的最大差异
float matrix_diff(float* a, float* b, int rows, int cols) {
    float max_diff = 0.0f;
    
    for (int i = 0; i < rows * cols; i++) {
        float diff = fabs(a[i] - b[i]);
        if (diff > max_diff) {
            max_diff = diff;
        }
    }
    
    return max_diff;
}

// 计算执行时间 (毫秒)
double calculate_execution_time(clock_t start, clock_t end) {
    return ((double)(end - start) / CLOCKS_PER_SEC) * 1000.0;
}

int main() {
    // 设置控制台代码页为简体中文GBK，以支持中文显示
    #ifdef _WIN32
    system("chcp 936 > nul");
    #endif
    
    printf("=== CUDA卷积计算程序 ===\n\n");
    
    // 设置矩阵尺寸
    int inputRows = 8, inputCols = 8;
    int kernelRows = 3, kernelCols = 3;
    int outputRows = inputRows - kernelRows + 1;
    int outputCols = inputCols - kernelCols + 1;
    
    // 输出CUDA设备信息
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    CUDA_CHECK_ERROR(err);
    
    printf("检测到 %d 个CUDA设备\n", deviceCount);
    
    hipDeviceProp_t deviceProp;
    for (int i = 0; i < deviceCount; i++) {
        err = hipGetDeviceProperties(&deviceProp, i);
        CUDA_CHECK_ERROR(err);
        
        printf("设备 %d: %s\n", i, deviceProp.name);
        printf("  计算能力: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  多处理器数量: %d\n", deviceProp.multiProcessorCount);
        printf("  全局内存: %.2f GB\n", deviceProp.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
        printf("  最大线程数/块: %d\n", deviceProp.maxThreadsPerBlock);
        printf("  最大线程数/多处理器: %d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf("  时钟频率: %.2f GHz\n\n", deviceProp.clockRate / 1000000.0f);
    }
    
    // 分配主机内存
    float* h_input = allocate_matrix_cpu(inputRows, inputCols);
    float* h_kernel = allocate_matrix_cpu(kernelRows, kernelCols);
    float* h_output_cuda = allocate_matrix_cpu(outputRows, outputCols);
    float* h_output_cpu = allocate_matrix_cpu(outputRows, outputCols);
    float* h_output_shared = allocate_matrix_cpu(outputRows, outputCols);
    
    // 生成随机输入矩阵
    create_random_matrix(h_input, inputRows, inputCols, 0.0f, 10.0f);
    
    // 创建边缘检测卷积核
    create_edge_detection_kernel(h_kernel);
    
    // 打印输入矩阵和卷积核
    print_matrix(h_input, inputRows, inputCols, "输入矩阵");
    print_matrix(h_kernel, kernelRows, kernelCols, "卷积核 (拉普拉斯算子)");
    
    // === CPU卷积计算 ===
    clock_t cpu_start = clock();
    convolve_cpu(h_input, h_kernel, h_output_cpu, inputRows, inputCols, kernelRows, kernelCols);
    clock_t cpu_end = clock();
    double cpu_time = calculate_execution_time(cpu_start, cpu_end);
    
    // === CUDA卷积计算 (基础版本) ===
    clock_t cuda_start = clock();
    convolve_cuda(h_input, h_kernel, h_output_cuda, inputRows, inputCols, kernelRows, kernelCols, false);
    clock_t cuda_end = clock();
    double cuda_time = calculate_execution_time(cuda_start, cuda_end);
    
    // === CUDA卷积计算 (共享内存优化版本) ===
    clock_t cuda_shared_start = clock();
    convolve_cuda(h_input, h_kernel, h_output_shared, inputRows, inputCols, kernelRows, kernelCols, true);
    clock_t cuda_shared_end = clock();
    double cuda_shared_time = calculate_execution_time(cuda_shared_start, cuda_shared_end);
    
    // 打印结果
    print_matrix(h_output_cpu, outputRows, outputCols, "CPU卷积结果");
    print_matrix(h_output_cuda, outputRows, outputCols, "CUDA卷积结果 (基础版本)");
    print_matrix(h_output_shared, outputRows, outputCols, "CUDA卷积结果 (共享内存版本)");
    
    // 验证结果
    float diff_cpu_cuda = matrix_diff(h_output_cpu, h_output_cuda, outputRows, outputCols);
    float diff_cpu_shared = matrix_diff(h_output_cpu, h_output_shared, outputRows, outputCols);
    
    printf("CPU与CUDA基础版本的最大差异: %f\n", diff_cpu_cuda);
    printf("CPU与CUDA共享内存版本的最大差异: %f\n", diff_cpu_shared);
    
    // 输出性能比较
    printf("\n=== 性能比较 ===\n");
    printf("CPU执行时间: %.2f 毫秒\n", cpu_time);
    printf("CUDA基础版本执行时间: %.2f 毫秒 (加速比: %.2fx)\n", 
           cuda_time, cpu_time / cuda_time);
    printf("CUDA共享内存版本执行时间: %.2f 毫秒 (加速比: %.2fx)\n", 
           cuda_shared_time, cpu_time / cuda_shared_time);
    
    // 尝试测试更大规模的矩阵
    printf("\n=== 大规模矩阵测试 ===\n");
    
    // 释放之前的内存
    free(h_input);
    free(h_kernel);
    free(h_output_cuda);
    free(h_output_cpu);
    free(h_output_shared);
    
    // 更大的测试矩阵
    inputRows = 1024;
    inputCols = 1024;
    kernelRows = 5;
    kernelCols = 5;
    outputRows = inputRows - kernelRows + 1;
    outputCols = inputCols - kernelCols + 1;
    
    // 重新分配内存
    h_input = allocate_matrix_cpu(inputRows, inputCols);
    h_kernel = allocate_matrix_cpu(kernelRows, kernelCols);
    h_output_cuda = allocate_matrix_cpu(outputRows, outputCols);
    h_output_cpu = allocate_matrix_cpu(outputRows, outputCols);
    h_output_shared = allocate_matrix_cpu(outputRows, outputCols);
    
    // 生成随机大矩阵和高斯卷积核
    create_random_matrix(h_input, inputRows, inputCols, 0.0f, 1.0f);
    create_gaussian_kernel(h_kernel, kernelRows, 1.0f);
    
    printf("大规模测试: 输入矩阵 %dx%d, 卷积核 %dx%d\n", inputRows, inputCols, kernelRows, kernelCols);
    
    // === CPU卷积计算 (大规模) ===
    cpu_start = clock();
    convolve_cpu(h_input, h_kernel, h_output_cpu, inputRows, inputCols, kernelRows, kernelCols);
    cpu_end = clock();
    cpu_time = calculate_execution_time(cpu_start, cpu_end);
    
    // === CUDA卷积计算 (基础版本, 大规模) ===
    cuda_start = clock();
    convolve_cuda(h_input, h_kernel, h_output_cuda, inputRows, inputCols, kernelRows, kernelCols, false);
    cuda_end = clock();
    cuda_time = calculate_execution_time(cuda_start, cuda_end);
    
    // === CUDA卷积计算 (共享内存优化版本, 大规模) ===
    cuda_shared_start = clock();
    convolve_cuda(h_input, h_kernel, h_output_shared, inputRows, inputCols, kernelRows, kernelCols, true);
    cuda_shared_end = clock();
    cuda_shared_time = calculate_execution_time(cuda_shared_start, cuda_shared_end);
    
    // 验证结果
    diff_cpu_cuda = matrix_diff(h_output_cpu, h_output_cuda, outputRows, outputCols);
    diff_cpu_shared = matrix_diff(h_output_cpu, h_output_shared, outputRows, outputCols);
    
    printf("大规模测试 - CPU与CUDA基础版本的最大差异: %f\n", diff_cpu_cuda);
    printf("大规模测试 - CPU与CUDA共享内存版本的最大差异: %f\n", diff_cpu_shared);
    
    // 输出大规模测试性能比较
    printf("\n=== 大规模测试性能比较 ===\n");
    printf("CPU执行时间: %.2f 毫秒\n", cpu_time);
    printf("CUDA基础版本执行时间: %.2f 毫秒 (加速比: %.2fx)\n", 
           cuda_time, cpu_time / cuda_time);
    printf("CUDA共享内存版本执行时间: %.2f 毫秒 (加速比: %.2fx)\n", 
           cuda_shared_time, cpu_time / cuda_shared_time);
    
    // 释放内存
    free(h_input);
    free(h_kernel);
    free(h_output_cuda);
    free(h_output_cpu);
    free(h_output_shared);
    
    // 重置CUDA设备
    hipDeviceReset();
    
    printf("\n程序执行完毕\n");
    
    return 0;
} 